#include "hip/hip_runtime.h"
﻿/*
	CUDA kernel for Linear SVM solver based on LIBLINEAR package
	 http://www.csie.ntu.edu.tw/~cjlin/liblinear/
	 Paper: "A Dual Coordinate Descent Method for Large-scale Linear SVM" Hsieh et al., ICML 2008

	

*/



//texture for vector, which is used for matrix vector multiplication
//in SVM, when we have to compute many dot products (one vector with others)
texture<float,1,hipReadModeElementType> mainVectorTexRef;


texture<float,1,hipReadModeElementType> deltasTexRef;

//texture fo labels assiociated with vectors
texture<float,1,hipReadModeElementType> labelsTexRef;


//constant array for diagonal shift in L2-SVM diag_shift[]={ 0.5/Cn, 0 , 0.5/Cp}
//where Cn,Cp penalty parameters for negative elements and positive
__device__  __constant__ float diag_shift[3];

//BB step for updateing alpha
__device__ __constant__ float stepBB=0.0f;


//main vector dimension
__device__ __constant__ int Dim;


// 1/square(Dim)
__device__ __constant__ float stepScaling=0.0f;

#define BLOCK_SIZE 128

#define WARP_SIZE 32



/*
	function checks if x is positive without 'if' statement

	if x> 0 return 1
	else return 0
*/
__device__ int isPositive(float x)
{ 
	//signbit returns 1 if x is negative and 0 otherwise
	// could be a problem if x=-0.0 ?
/* 
 int pos = signbit(x);	//  0-if x>0	1 if x<0	0 if x=0
  int neg = signbit(-x);//  1-if x>0	0 if x<0	0 if x=0
  
  return neg*(1-pos);
*/
  //other solution
 
  float test = x>0.0f;
  return 1.0f &&test;
  
}



/*
Based on cuda kernels from 
"Efcient Sparse Matrix-Vector Multiplication on CUDA" Nathan Bell and Michael Garlandy
December 11, 2008
*/
//
//cuda kernel funtion for computing part of Gradient in method of solving linear SVM,
//grad = w'*xi*yi-1+alpha[i]*C
//this cuda kernel computes only first part w'*xi*yi where w-vector is in tex cache, yi is in tex cache
// xi - is i-th row in matrix containning all elements, matrix is in CSR fromat
//Remarks: based on spmv_csr_vector_kernel from publication above
//Params:
//vals - array of vectors values
//idx  - array of vectros indexes in CSR fromat
//vecPointers -array of pointers(indexes) to idx and vals array to specific vectors
//results - array of results Linear Kernel
//num_rows - number of vectors, stored in CSR matrix format, each vector is stored in one row of matrix

extern "C" __global__ void ComputeDotProd(const float * vals,
									   const int * idx, 
									   const int * vecPointers, 
									   float * results,
									   const int num_rows)
{
	__shared__ float sdata[BLOCK_SIZE + 16];                          // padded to avoid reduction ifs
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];
		

	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	for(int row = warp_id; row < num_rows; row += num_warps){
		// use two threads to fetch vecPointers[row] and vecPointers[row+1]
		// this is considerably faster than the straightforward version
		if(thread_lane < 2)
			ptrs[warp_lane][thread_lane] = vecPointers[row + thread_lane];
		const int row_start = ptrs[warp_lane][0];                   //same as: row_start = vecPointers[row];
		const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = vecPointers[row+1];

		// compute local sum
		float sum = 0;
		for(int jj = row_start + thread_lane; jj < row_end; jj += WARP_SIZE)
			sum += vals[jj] * tex1Dfetch(mainVectorTexRef,idx[jj]-1); //all indexes starts from 1, but mainVector starts from 0

		// reduce local sums to row sum (ASSUME: warpsize 32)
		sdata[threadIdx.x] = sum;
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads(); 
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();
	   

		

		// first thread writes warp result
		if (thread_lane == 0){
			
			results[row] = tex1Dfetch(labelsTexRef,row)*sdata[threadIdx.x];
		}

			
	}//end for
}


/*
	Finalize computing gradient for l2 svm formulation
	grad= w'*x_i*y_i-1+alpha_i*diag 

	Params:
	partGrad - in/out parameter, 
*/
extern "C" __global__ void GradientFinalize(float * partGrad,
											float* alpha,
											const int size)
{

	int i =  blockDim.x * blockIdx.x + threadIdx.x;

	if(i>=size){
		return;	
	}
	
	
	float grad =partGrad[i];
	
	float yi = tex1Dfetch(labelsTexRef,i);
	float alpha_i = alpha[i];

	//in LibLinear we have to compute
	//G=W*element*yi-1+alpah[i]*Dii
	grad = grad-1;
	
	grad+=alpha_i*diag_shift[(int)yi+1];
	
/*  
	!!!!!!!!!!!!1 Uwaga testowo mnoże przez yi, normalnie nie powinno tego być
*/
	partGrad[i]=grad;
	
/*  Projected gradient
	float PG=grad;
	
	int signG = signbit(PG);
	int isPosAlpha = isPositive(alpha_i);
	PG=PG*ceilf((signG+isPosAlpha+0.0f)/(signG+isPosAlpha+1.0f));

	//if PG< 1e-12, to znaczy że już jesteśmy w optimum,
	//lecz to powinno zachodzić dla wszystkich
	//we store 
	partGrad[i]=PG;
*/
	
}


/*
	Update alpha by step*grad
	alpha= alpha-step*grad;
	deltas= alpha_new - alpha_old

	step is copied into device constatnt "stepBB"

*/
extern "C" __global__ void UpdateAlpha(const float * grad,
									   float* alpha,
									   float* deltas,
									   const int size)
{

	int i =  blockDim.x * blockIdx.x + threadIdx.x;

	if(i>=size){
		return;	
	}

	float old_alpha = alpha[i];

	//stepBB is module constant, see at beginning this file
	float new_alpha = fmaxf(old_alpha+ stepBB*grad[i],0.0f);

	deltas[i]=(new_alpha-old_alpha)*tex1Dfetch(labelsTexRef,i);
	alpha[i] = new_alpha;
}


/*
	Computes vector square norm using parallel reduction
	norm = vec'*vec
	This kernel is needed for computing objective function value
	obj =0.5*[ w*w+ alpha'*(C*alpha-2)]
	and compute second part alpha'*(C*alpha-2)

	w- aray
	reducted - array for reducted results
	n - size of w array

*/
extern "C" __global__ void VectorSquareW(float * w, float* reducted, const int n)
{
	__shared__ float sdata[BLOCK_SIZE + 16];        
	

// perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    //unsigned int i = blockIdx.x*BLOCK_SIZE*2 + threadIdx.x;
    //unsigned int gridSize = BLOCK_SIZE*2*gridDim.x;

	unsigned int blockSize = blockDim.x;
	unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    float w_i=0;
	while (i < n)
    {   
		w_i = w[i];
      
        mySum += w_i*w_i;
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (i + blockSize < n) {
			w_i= w[i+blockSize];
            mySum += w_i*w_i;  
		}
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float* smem = sdata;
        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; __syncthreads(); }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; __syncthreads(); }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; __syncthreads(); }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; __syncthreads(); }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; __syncthreads(); }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; __syncthreads(); }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        reducted[blockIdx.x] = sdata[0];
	
}


/*
	Computes alpha square part in computing value for objective function using parallel reduction

	This kernel is needed for computing objective function value
	obj =0.5*[ w*w+ alpha'*(C*alpha-2)]
	and compute second part alpha'*(C*alpha-2)

*/
extern "C" __global__ void VectorSquareAlpha(float * alpha,float* reducted, const int n)
{

__shared__ float sdata[BLOCK_SIZE + 16];        
	

// perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    //unsigned int i = blockIdx.x*BLOCK_SIZE*2 + threadIdx.x;
    //unsigned int gridSize = BLOCK_SIZE*2*gridDim.x;

	unsigned int blockSize = blockDim.x;
	unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;
    
    float mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    float alhpa_i=0;
	float yi=0;
	while (i < n)
    {   
		alhpa_i = alpha[i];
		yi = tex1Dfetch(labelsTexRef,i);
      
        mySum += alhpa_i*(alhpa_i* diag_shift[(int)yi+1]-2);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (i + blockSize < n) {
            alhpa_i = alpha[i+blockSize];
			yi = tex1Dfetch(labelsTexRef,i+blockSize);      
			mySum += alhpa_i*(alhpa_i* diag_shift[(int)yi+1]-2);
		}
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float* smem = sdata;
        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; __syncthreads(); }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; __syncthreads(); }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; __syncthreads(); }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; __syncthreads(); }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; __syncthreads(); }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; __syncthreads(); }
    }
    
    // write result for this block to global mem 
    if (tid == 0) 
        reducted[blockIdx.x] = sdata[0];

	
}

extern "C" __global__ void ComputeLinPart(float * alpha,float* alphaOld,float* grad, float* reducted, const int size)
{


	
}


/*
	Computes BB steps using parallel reduction

	step1 = (x_new-x_old)'*(x_new-x_old)/ (x_new-x_old)'*(grad_new - grad_old)
	step2 = (grad_new - grad_old)'*(grad_new - grad_old)/ (x_new-x_old)'*(grad_new - grad_old)
*/
 extern "C" __global__ void ComputeBBSteps(const float * alpha, 
											const float* alpha_old,
											const float* grad,
											const float* grad_old,
											float* reductedAlphaPart,
											float* reductedGradPart,
											float* reductedAlphaGradPart,
											const int size)
{


	
}



//cuda kernel funtion for updating  W-vector in method of solving linear SVM,
//the idea is almost the same as in CudaDotProd function, 
//each warp computes multiplication between step vector (D) and each column
//
//
//					   | x11 x12 .. x1n|
//					   | x21 x22 .. x2n|
//	[D1, D2, ..., Dl]* | .    .  ..  . |
//					   | .    .  ..  . |
//					   | xl1 xl2 .. xln|
// l- number of elements
// n - vector dim
// we have to compute sums  sum_k = Sum_i (D_i*x_ik)
// sum_1 = D1*x11+ D2*x21 +...+Dl*xl1
// sum_2 =
// ...
// sum_l
// when we have sums we can compute change for vector W
// W[k]+= sum_k
//
//matrix is in CSC fromat
//Params:
//vals - array of vectors values, column order
//idx  - array of vectros indexes in CSC fromat (compact sparse column)
//vecPointers -array of pointers(indexes) to idx and vals array, indicates start and end of specific column
//W - computed W vector - array of size dim, 
//num_cols - number of vectors, stored in CSC matrix format, 
extern "C" __global__ void update_W(const float * vals,
									   const int * idx, 
									   const int * vecPointers, 
									   float * W,
									   const int num_rows)
{

//todo: change all  "*rows" into columns
	__shared__ float sdata[BLOCK_SIZE + 16];                          // padded to avoid reduction ifs
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];
		

	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	for(int row = warp_id; row < num_rows; row += num_warps){
		// use two threads to fetch vecPointers[row] and vecPointers[row+1]
		// this is considerably faster than the straightforward version
		if(thread_lane < 2)
			ptrs[warp_lane][thread_lane] = vecPointers[row + thread_lane];
		const int row_start = ptrs[warp_lane][0];                   //same as: row_start = vecPointers[row];
		const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = vecPointers[row+1];

		// compute local sum
		float sum = 0;
		for(int jj = row_start + thread_lane; jj < row_end; jj += WARP_SIZE)
			sum += vals[jj] * tex1Dfetch(deltasTexRef,idx[jj]); //deltas was already mul by yi in prev kernel

		// reduce local sums to row sum (ASSUME: warpsize 32)
		sdata[threadIdx.x] = sum;
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads(); 
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();
	   

		

		// first thread writes warp result
		if (thread_lane == 0){
			
			//results[row] = tex1Dfetch(labelsTexRef,row)*sdata[threadIdx.x];
			W[row] +=sdata[threadIdx.x];
		}

			
	}
}






/*
	Implements solve_l2r_l1l2_svc method.
	Cuda kernel computes outer loop in main algorithm.
	Elements matrix is in CSR format
	[vector values]
	[vector indexes]
	[pointers to starting index to vector "i"]

	N - number of objects for classification
	L - number of object features 
Params:

QD  - array of size N, diagonal cache QD=Qii+diag
diag - 3 dim array, diag[] = { 0.5/Cneg, 0, 0.5/Cpos} , specific diag is taken by diag[yi+1]
	   if yi=-1 we take diag[0], if yi=1 we take diag[2], diag[1] - is not used
alpha - array with alpha coeficients of size N
paramsC - 
G - array of size N,contains precomputed dot product between vector "W" and all obcjects(vectors) multipicated by label, 
	after computation G stores al projected gradient for chcecking stop creterion, this is  "in out" parameter
deltas - array of size N, contains step in each dimension, out parameter

*/
extern "C" __global__ void lin_l2r_l2_svc_solver_with_gradient(
	const float* QD,
	float* alpha,
	float* G,
	float* deltas,
	const int elements
	)
{

	int i =  blockDim.x * blockIdx.x + threadIdx.x;
	//grad = W'* element[i]*Y[i]

	if(i<elements){
	
	float grad = G[i];
	
	float yi = tex1Dfetch(labelsTexRef,i);
	float alpha_i = alpha[i];

	//in LibLinear we have to compute
	//G=W*element*yi-1+alpah[i]*Dii
	grad = grad-1;
	
	grad+=alpha_i*diag_shift[(int)yi+1];

	float PG=0;
	
	/*
	below we compute projected gradient, but we don't want use 'if' statemets
		if alpha[i]==0
			PG=min(0,grad[i])
		else if alpha[i]==C
			PG=max(0,grad[i]
		else
			PG=grad[i]
	
	we use formula:
	1. map alpha[i] to 
		-1 - alpha[i]==0
		 0   0<alpha[i]<C
		 1   alpah[i]==C

		 mapAlpha = -1+ floor(alpha[i]/C)+ceil(alpha[i]/C);
		 
		 what if C==infinity? 

	2. compute PG base on maped alpha
	pg= pg+floor(0.5*sign(pg*mapAlpha))*pg
	
	float alpha_C = alpha[i]/C;
	int mapAlpha= -1+floorf(alpha_C)+ceil(alpha_C);
	PG= PG+floor(0.5*sign(PG*mapAlpha))*PG;

	G[i] = PG;
	*/
	/*
	for L2-SVM we can simplify expresion, we don't have to check if alpha[i]=C because C is infinity
	*/
	PG=grad;
	

	/*
	this computes PG without using 'if' statements, line 472 in lin solver

	

	if alpha_i=0
		PG= min(0,grad)
	else
		PG=grad

	we could change this to
	if alpha_i=0 and grad <0
		PG = grad
	if alpha_i=0 and grad >=0
		PG = 0
	if alpha_i>0 and grad <0
		PG = grad
	if alpha_i=> and grad >=0
		PG = grad

	we can set PG using formula:
	signG=1 if grad<0
	signG=0 if grad>=0

	isPosAlpha=1 if alpha_i>0
	isPosAlpha=0 if alpha_i<=0

	float ifTest=(signG+isPosAlpha+0.0f)/(signG+isPosAlpha+1.0f);
	ifTest= ceilf(ifTest);
	PG=ifTest*PG;
	*/
	int signG = signbit(PG);
	int isPosAlpha = isPositive(alpha_i);
	PG=PG*ceilf((signG+isPosAlpha+0.0f)/(signG+isPosAlpha+1.0f));

	//if PG< 1e-12, to znaczy że już jesteśmy w optimum,
	//lecz to powinno zachodzić dla wszystkich
	//we store 
	
	G[i]=PG;
	//G[i]=diag_shift[(int)yi+1]-5;

	//we should compute delta only if PG>0
	//but we want to omit branching so we computed delta but
	//grad = PG  if PG=0 then delta ==0
	grad=PG;
	
//normaly in paper is Min(Max(alpha-G/QD[i],0.0),U) but in our case U is infinty 
	//so min part was ommitted
	//float deltaAlpha = fmaxf(alpha_i-grad/(QD[i]+diag_shift[(int)yi+1] ),0.0f)-alpha_i;

	float deltaAlpha = fmaxf(alpha_i- stepScaling*grad,0.0f)-alpha_i;

	
	//stepScaling - scaling parameter
	//deltas[i]=stepScaling*deltaAlpha*yi;
	//set new alpha
	//alpha[i]=alpha_i+deltaAlpha*stepScaling;

	deltas[i]=yi*deltaAlpha;
	alpha[i]=alpha_i+deltaAlpha;
}//end if(i<elements)

}