#include "hip/hip_runtime.h"
﻿



texture<float,1,hipReadModeElementType> mainVecTexRef;

//texture fo labels assiociated with vectors
texture<float,1,hipReadModeElementType> labelsTexRef;

__device__ const int ThreadPerRow=4;
__device__ const int SliceSize=64;
//gamma parameter in RBF
//__constant__ float GammaDev=0.5;

extern __shared__  float sh_data[];

//Use sliced Ellpack format for computing rbf kernel
//vecVals - vectors values in Sliced Ellpack,
//vecCols - array containning column indexes for non zero elements
//vecLengths  - number of non zero elements in row
//sliceStart   - determine where particular slice starts and ends
//selfDot    - precomputed self dot product
//result  - for final result
//mainVecIdx - index of main vector
//nrows   - number of rows
//ali	  - align
extern "C" __global__ void rbfSlicedEllpackKernel(const float *vecVals,
											 const int *vecCols,
											 const int *vecLengths, 
											 const int * sliceStart, 
											 const float* selfDot,
											 const float* vecLabels,
											 float *result,
											 const int mainVecIdx,
											 const int nrRows,
											 const float gamma, 
											 const int align){
  
  //sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
  //float* sh_cache = (float*)sh_data;
	__shared__  float sh_cache[ThreadPerRow*SliceSize];
	
	__shared__ int shMainVecIdx;
	__shared__ float shMainSelfDot;
	__shared__ float shLabel;
	__shared__ float shGamma;
	
	if(threadIdx.x==0)
	{
		shMainVecIdx=mainVecIdx;
		shMainSelfDot = selfDot[shMainVecIdx];
		shLabel = vecLabels[shMainVecIdx];
		shGamma=gamma;
	}

  int tx = threadIdx.x;
  int txm = tx % 4; //tx% ThreadPerRow
  int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);
  
  //map group of thread to row, in this case 4 threads are mapped to one row
  int row =  thIdx>> 2; // 
  
  if (row < nrRows){
	  float sub = 0.0;
	   int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
	  
	  for(int i=0; i < maxRow; i++){
		  int ind = i*align+sliceStart[blockIdx.x]+tx;
		  float value = vecVals[ind];
		  int col     = vecCols[ind];
		  sub += value * tex1Dfetch(mainVecTexRef, col);
	  }
  
   sh_cache[tx] = sub;
   
   //for 4 thread per row
  
if(txm < 2){
	  sh_cache[tx]+=sh_cache[tx+2];
	  sh_cache[tx] += sh_cache[tx+1];

	  if(txm == 0 ){
		  result[row]=vecLabels[row]*shLabel*expf(-0.5*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
		//result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
		  //result[row]=vecLabels[row]*shLabel*expf(-GammaDev*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
	  }
   }


}//if row<nrRows 

/*
if(txm < 2){
	  sh_cache[tx]+=sh_cache[tx+2];
	  sh_cache[tx] += sh_cache[tx+1];
}

if(thIdx<nrRows ){

  result[thIdx]=vecLabels[thIdx]*shLabel*expf(-shGamma*(selfDot[thIdx]+shMainSelfDot-2*sh_cache[4*tx]));
}*/

  
}//end func


