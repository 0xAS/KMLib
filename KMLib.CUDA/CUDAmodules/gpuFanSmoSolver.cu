#include "hip/hip_runtime.h"
﻿
#include <float.h>

const float MAX_FLOAT = FLT_MAX;
const float MIN_FLOAT = FLT_MIN;


__constant__ float C;
// minimal coeficient
__constant__ float COEF_EPS = 0.00001f;

// constat for kernel diagonal for index i
//__constant__ float QD_i;

// label for i-th example
//__constant__ float Y_i;

//texture for vector, which is used for matrix vector multiplication
//in SVM, when we have to compute many dot products (one vector with others)
texture<float,1,hipReadModeElementType> mainVectorTexRef;


#define BLOCK_SIZE 128

#define WARP_SIZE 32

#define NEG_INFINITY_F __int_as_float(0xff800000)



/*

	Do warp parallel reduction in order to find max value and its index
*/
__device__ void maxWarpReduce(volatile int *volShIdx,volatile float *volShVal,unsigned int tid)
{
		if (BLOCK_SIZE >=  64) { if( volShVal[tid]< volShVal[tid+32]) {
							 volShVal[tid]=volShVal[tid+32]; volShIdx[tid]=volShIdx[tid+32];	} }
		if (BLOCK_SIZE >=  32) { if( volShVal[tid]< volShVal[tid+16]) {
							 volShVal[tid]=volShVal[tid+16]; volShIdx[tid]=volShIdx[tid+16];	} }
		if (BLOCK_SIZE >=  16) { if( volShVal[tid]< volShVal[tid+8]) {
							 volShVal[tid]=volShVal[tid+8]; volShIdx[tid]=volShIdx[tid+8];	} }
		if (BLOCK_SIZE >=   8) { if( volShVal[tid]< volShVal[tid+4]) {
							 volShVal[tid]=volShVal[tid+4]; volShIdx[tid]=volShIdx[tid+4];	} }
		if (BLOCK_SIZE >=   4) { if( volShVal[tid]< volShVal[tid+2]) {
							 volShVal[tid]=volShVal[tid+2]; volShIdx[tid]=volShIdx[tid+2];	} }
		if (BLOCK_SIZE >=   2) { if( volShVal[tid]< volShVal[tid+1]) {
							 volShVal[tid]=volShVal[tid+1]; volShIdx[tid]=volShIdx[tid+1];	} }
}

/*
  Do warp parallel reduction for minimum finding
*/
__device__ void minWarpReduce(volatile float *sdata,unsigned int tid)
{
		if (BLOCK_SIZE >=  64) sdata[tid]=fminf(sdata[tid],sdata[tid+32]);
		if (BLOCK_SIZE >=  32) sdata[tid]=fminf(sdata[tid],sdata[tid+16]);
		if (BLOCK_SIZE >=  16) sdata[tid]=fminf(sdata[tid],sdata[tid+8]);
		if (BLOCK_SIZE >=   8) sdata[tid]=fminf(sdata[tid],sdata[tid+4]);
		if (BLOCK_SIZE >=   4) sdata[tid]=fminf(sdata[tid],sdata[tid+2]);
		if (BLOCK_SIZE >=   2) sdata[tid]=fminf(sdata[tid],sdata[tid+1]);
}

/*
	Do parallel reduction for finding index "i" which maximize
	// i: Maximizes -y_i * grad(f)_i, i in I_up(\alpha)
*/
extern "C" __global__ void FindMaxIdx(const float* y, 
									  const float* alpha, 
									  const float* grad,
									  int * idxReduce, 
									  float* gradReduce,
									  const int N)
{

	__shared__ float shVals[BLOCK_SIZE];     
	__shared__ int shIdx[BLOCK_SIZE];
	
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	//unsigned int i = blockIdx.x*BLOCK_SIZE*2 + threadIdx.x;
	//unsigned int gridSize = BLOCK_SIZE*2*gridDim.x;

	unsigned int blockSize = blockDim.x;
	unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
	unsigned int gridSize = blockDim.x*2*gridDim.x;
	
	shVals[tid]=NEG_INFINITY_F;
   
	// we reduce multiple elements per thread.  The number is determined by the 
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	float maxG=NEG_INFINITY_F ;
	float tempMax=NEG_INFINITY_F ;
	float yi=0;
	float alpha_i=0;
	float grad_i=0;
	
	while (i < N)
	{   
		yi=y[i];
		alpha_i=alpha[i];
		tempMax = (yi*alpha_i)<(yi==1?C:0) ? -(grad[i]*yi):NEG_INFINITY_F;
		maxG = fmaxf(maxG, tempMax );
		//if maxG==tempMax then tempMax is new max value, so remember its index, otherwise do nothing (return 0)
		maxG==tempMax ? shIdx[tid]=i:0; 

		// ensure we don't read out of bounds 
		if (i + blockSize < N) {
			yi=y[i + blockSize];
			alpha_i=alpha[i + blockSize];
			tempMax = (yi*alpha_i)<(yi==1?C:0) ? -(grad[i + blockSize]*yi):NEG_INFINITY_F;
			maxG = fmaxf(maxG, tempMax );
			//if maxG==tempMax then tempMax is new max value, so remember its index, otherwise do nothing (return 0)
			maxG==tempMax ? shIdx[tid]=i+blockSize:0; 
		}
		i += gridSize;
	} 

	// each thread puts its local sum into shared memory 
	shVals[tid] = maxG;
	__syncthreads();


	// do reduction in shared mem
	if (BLOCK_SIZE >= 512) { 
		if (tid < 256) { if( shVals[tid]< shVals[tid+256]) {
							 shVals[tid]=shVals[tid+256]; shIdx[tid]=shIdx[tid+256];	}} __syncthreads(); }
	if (BLOCK_SIZE >= 256) { 
		if (tid < 128) { if( shVals[tid]< shVals[tid+128]) {
							 shVals[tid]=shVals[tid+128]; shIdx[tid]=shIdx[tid+128];	}} __syncthreads(); }
	if (BLOCK_SIZE >= 128) { 
		if (tid < 64) { if( shVals[tid]< shVals[tid+64]) {
							 shVals[tid]=shVals[tid+64]; shIdx[tid]=shIdx[tid+64];	}} __syncthreads(); }
	

	if (tid < 32)
	{
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		maxWarpReduce(shIdx,shVals,tid);
	}
	
	// write result for this block to global mem 
	if (tid == 0) {
		gradReduce[blockIdx.x] = shVals[0];
		idxReduce[blockIdx.x] = shIdx[0];
	}
	
}


/*
	Do parallel reduction for finding index "j" which minimaze
	j: mimimizes the decrease of obj value
    (if quadratic coefficeint <= 0, replace it with tau)
    -y_j*grad(f)_j < -y_i*grad(f)_i, j in I_low(\alpha)

y - 
alpha - 
grad  - 
Qi    - i-th column in kernel matrix, each value was mul by yi*yj
*/
extern "C" __global__ void FindMinIdx(const float * y,		//labels 
									  const float* alpha,   //alpha coef
									  const float* grad,	// gradient
									  const float* Qi,		// i-th column in kernel matrix
									  const float* QD,		// diagonal in kernel matris
									  int * idxReduce,		// array for results
									  float* gradReduce,	// array for results
									  float GMax,
									  float QD_i,
									  float Y_i,
									  const int N)
{

	__shared__ float shVals[BLOCK_SIZE];     
	__shared__ int shIdx[BLOCK_SIZE];
	
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	//unsigned int i = blockIdx.x*BLOCK_SIZE*2 + threadIdx.x;
	//unsigned int gridSize = BLOCK_SIZE*2*gridDim.x;

	unsigned int blockSize = blockDim.x;
	unsigned int j = blockIdx.x*blockDim.x*2 + threadIdx.x;
	unsigned int gridSize = blockDim.x*2*gridDim.x;
	
	shVals[tid]=NEG_INFINITY_F;
   
	// we reduce multiple elements per thread.  The number is determined by the 
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	float maxG=NEG_INFINITY_F ;
	float tempMax=NEG_INFINITY_F ;
	float yj=0;
	float alpha_j=0;
	float grad_j=0;
	float quad_coef=0;
	
	while (j < N)
	{   
		yj=y[j];
		alpha_j=alpha[j];
		//in libsvm this formula is different for different value of yj, 
		//but when we mul by labels for i,j examples this formula can be computed as below
		quad_coef = (QD_i+QD[j]-2*Y_i*yj*Qi[j]);

		//check if is not at lower bound
		tempMax = (yj*alpha_j)>(yj==1?0:-C) ? __fdividef(__powf(GMax+yj*grad[j],2.f),quad_coef):NEG_INFINITY_F;
		maxG = fmaxf(maxG, tempMax );
		//if maxG==tempMax then tempMax is new max value, so remember its index, otherwise do nothing (return 0)
		maxG==tempMax ? shIdx[tid]=j:0; 
		//atomicMax, atomicCLA??

		// ensure we don't read out of bounds 
		if (j + blockSize < N) {
			yj=y[j + blockSize];
			alpha_j=alpha[j + blockSize];
			quad_coef = (QD_i+QD[j+ blockSize]-2*Y_i*yj*Qi[j+ blockSize]);
			tempMax = (yj*alpha_j)>(yj==1?0:-C) ? __fdividef(__powf(GMax+yj*grad[j+blockSize],2.f),quad_coef):NEG_INFINITY_F;
			maxG = fmaxf(maxG, tempMax );
			//if maxG==tempMax then tempMax is new max value, so remember its index, otherwise do nothing (return 0)
			maxG==tempMax ? shIdx[tid]=j+blockSize:0; 
		}
		j+= gridSize;
	} 

	// each thread puts its local sum into shared memory 
	shVals[tid] = maxG;
	__syncthreads();


	// do reduction in shared mem
	if (BLOCK_SIZE >= 512) { 
		if (tid < 256) { if( shVals[tid]< shVals[tid+256]) {
							 shVals[tid]=shVals[tid+256]; shIdx[tid]=shIdx[tid+256];	}} __syncthreads(); }
	if (BLOCK_SIZE >= 256) { 
		if (tid < 128) { if( shVals[tid]< shVals[tid+128]) {
							 shVals[tid]=shVals[tid+128]; shIdx[tid]=shIdx[tid+128];	}} __syncthreads(); }
	if (BLOCK_SIZE >= 128) { 
		if (tid < 64) { if( shVals[tid]< shVals[tid+64]) {
							 shVals[tid]=shVals[tid+64]; shIdx[tid]=shIdx[tid+64];	}} __syncthreads(); }
	

	if (tid < 32)
	{
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		maxWarpReduce(shIdx,shVals,tid);
		
	}
	
	// write result for this block to global mem 
	if (tid == 0) {
		gradReduce[blockIdx.x] = shVals[0];
		idxReduce[blockIdx.x] = shIdx[0];
	}
	
}



/*
	Finds min Gradient value for stopping criterion
*/
extern "C" __global__ void FindStoppingGradVal(const float* y, 
									  const float* alpha, 
									  const float* grad,
									  float* gradReduce,
									  const int N)
{
	__shared__ float shVals[BLOCK_SIZE];     
	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	//unsigned int i = blockIdx.x*BLOCK_SIZE*2 + threadIdx.x;
	//unsigned int gridSize = BLOCK_SIZE*2*gridDim.x;

	unsigned int blockSize = blockDim.x;
	unsigned int j = blockIdx.x*blockDim.x*2 + threadIdx.x;
	unsigned int gridSize = blockDim.x*2*gridDim.x;
	
	shVals[tid]=FLT_MAX;
	float yj=0;
	float yj_far=0;
	// we reduce multiple elements per thread.  The number is determined by the 
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread

	while (j < N)
	{   
		yj=y[j];
		yj_far = (j+BLOCK_SIZE)<N ? y[j+BLOCK_SIZE]:0;
		
		shVals[tid]=fminf(shVals[tid], fminf(
										(yj*alpha[j]) >(yj==1? 0:-C) ? -(grad[j]*yj): FLT_MAX,
										j+BLOCK_SIZE<N ?
										( ( yj_far*alpha[j+BLOCK_SIZE])>(yj_far==1? 0:-C) ? -(grad[j+BLOCK_SIZE]*yj_far): FLT_MAX)
										: FLT_MAX
										));
		
		j += gridSize;
	} 
	__syncthreads();


	// do reduction in shared mem
	if (BLOCK_SIZE >= 512) 
		if (tid < 256) { shVals[tid]=fminf(shVals[tid],shVals[tid+256]); __syncthreads(); }

	if (BLOCK_SIZE >= 256) 
		if (tid < 128) {  shVals[tid]=fminf(shVals[tid],shVals[tid+128]);  __syncthreads(); }

	if (BLOCK_SIZE >= 128)
		if (tid < 64) {  shVals[tid]=fminf(shVals[tid],shVals[tid+64]); __syncthreads(); }
	

	if (tid < 32)
	{
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		minWarpReduce(shVals,tid);
	}
	
	// write result for this block to global mem 
	if (tid == 0) {
		gradReduce[blockIdx.x] = shVals[0];
	}
	
}



/*

	Updates gradient 

	One threads process 4 gradients, inspired by Volkow http://www.cs.berkeley.edu/~volkov/volkov10-GTC.pdf
*/
extern "C" __global__ void UpdateGrad(const float* Qi, 
									  const float* Qj, 
									  float* grad,
									  float diff_i,
									  float diff_j,
									  const int N)
{
    int iblock = blockIdx.x+  gridDim.x*blockDim.x;
    int idx    = threadIdx.x+4*iblock*blockDim.x;
	//acumulators 
	float tempGrad[4];	
	float tempQi[4];	
	float tempQj[4];
	float alpha_i_diff=diff_i;
	float alpha_j_diff=diff_j;	
	//read 4 elements per thread int to register's
	for(int i=0;i<4;i++){
		tempGrad[i] = (idx+i*blockDim.x <N) ? grad[idx+i*blockDim.x]:0;
		tempQi[i]   = (idx+i*blockDim.x <N) ? Qi[idx+i*blockDim.x]:0;
		tempQj[i]   = (idx+i*blockDim.x <N) ? Qj[idx+i*blockDim.x]:0;

		//(idx+i*blockDim.x <N) ? (tempGrad[i] = grad[idx+i*blockDim.x]; tempQi[i]=Qi[idx+i*blockDim.x]; tempQj[i]=Qj[idx+i*blockDim.x]):0;
	}
	
	//do final computation
	for(int i=0;i<4;i++){
		(idx+i*blockDim.x <N) ? (grad[idx+i*blockDim.x]=tempGrad[i]+ alpha_i_diff*tempQi[i]+alpha_j_diff*tempQj[i]) :0;
		
	}
}