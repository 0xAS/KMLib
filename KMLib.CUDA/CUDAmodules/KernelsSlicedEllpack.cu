#include "hip/hip_runtime.h"
﻿/*
author: Krzysztof Sopyla
mail: krzysztofsopyla@gmail.com
License: MIT
web page: http://wmii.uwm.edu.pl/~ksopyla/projects/svm-net-with-cuda-kmlib/
*/

#include <float.h>

#include <Config.h>


#define ThreadPerRow 4
#define LOG_THREADS 2 // LOG2(ThreadPerRow)
#define SliceSize 64


template<int TexSel> __device__ void SpMV_SliceEllpack(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const int align,
	const int row,
	const int nrRows,
	volatile float* shDot);

//extern __shared__  float sh_data[];

//Use sliced Ellpack format for computing rbf kernel
//vecVals - vectors values in Sliced Ellpack,
//vecCols - array containing column indexes for non zero elements
//vecLengths  - number of non zero elements in row
//sliceStart   - determine where particular slice starts and ends
//selfDot    - precomputed self dot product
//result  - for final result
//mainVecIdx - index of main vector
//nrows   - number of rows
//align	  - align
extern "C" __global__ void rbfSlicedEllpackKernel(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfDot,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float shDot[ThreadPerRow*SliceSize];
		shDot[threadIdx.x]=0.0;	

		__shared__ int shMainVecIdx;
		__shared__ float shMainSelfDot;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfDot = selfDot[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
		}
		__syncthreads();
		
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);
		int txm = threadIdx.x %  ThreadPerRow;
		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> LOG_THREADS; // 

		if (row < nrRows){
			
			SpMV_SliceEllpack<1>(vecVals,vecCols,vecLengths,sliceStart,align,row,nrRows,shDot);
			if(txm == 0 ){
					result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*shDot[threadIdx.x]));
				}

		}//if row<nrRows 
}//end func




extern "C" __global__ void rbfSlicedEllpackKernel_old(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfDot,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float sh_cache[ThreadPerRow*SliceSize];

		__shared__ int shMainVecIdx;
		__shared__ float shMainSelfDot;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfDot = selfDot[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
		}

		int tx = threadIdx.x;
		int txm = tx %  ThreadPerRow;
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);

		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> LOG_THREADS; // 

		if (row < nrRows){
			float sub = 0.0;
			int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
			int col=-1;
			float value =0;
			int ind=0;

			for(int i=0; i < maxRow; i++){
				ind = i*align+sliceStart[blockIdx.x]+tx;
				col     = vecCols[ind];
				value = vecVals[ind];
				sub += value * tex1Dfetch(mainVecTexRef, col);
			}

			sh_cache[tx] = sub;
			__syncthreads();

			volatile float *shMem = sh_cache;


			for(int s=ThreadPerRow/2; s>0; s>>=1) //s/=2
			{
				if(txm < s){
					shMem[tx] += shMem[tx+s];
				}
			}

			if(txm == 0 ){
				result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
			}


			//for 4 thread per row
			//if(txm < 2){
			//	shMem[tx]+=shMem[tx+2];
			//	shMem[tx] += shMem[tx+1];
			//if(txm < 1){
			//	shMem[tx] += shMem[tx+1];
			//	if(txm == 0 ){
			//		result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
			//	}
			//}
		}//if row<nrRows 
}//end func





//TODO: impelmentacja rbfSlEll_ILP

//Use sliced Ellpack format for computing rbf kernel
//vecVals - vectors values in Sliced Ellpack,
//vecCols - array containning column indexes for non zero elements
//vecLengths  - number of non zero elements in row
//sliceStart   - determine where particular slice starts and ends
//selfDot    - precomputed self dot product
//result  - for final result
//mainVecIdx - index of main vector
//nrows   - number of rows
//ali	  - align
extern "C" __global__ void rbfSERTILP(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfDot,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float shDot[ThreadPerRow*SliceSize];

		//define ROWS_B BLOCK_SIZE/THREADS_ROW
		#define ROWS_B SliceSize
		//__shared__ int shMaxRows[ROWS_B];

		__shared__ int shMainVecIdx;
		__shared__ int shSliceStart;
		__shared__ float shMainSelfDot;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfDot = selfDot[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
			shSliceStart=sliceStart[blockIdx.x];
		}
		__syncthreads();

		//int tid = threadIdx.x;

		int idxT = threadIdx.x % ThreadPerRow; //thred number in Thread Goup
		int idxR = threadIdx.x/ThreadPerRow; //row index mapped into block region

		//int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);
		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  (blockIdx.x*blockDim.x+threadIdx.x)>> LOG_THREADS; 

		if (row < nrRows){

			
			//if(threadIdx.x < ROWS_B){
			//	unsigned int row2=blockIdx.x* ROWS_B+threadIdx.x;
			//	if(row2<nrRows){
			//		shMaxRows[threadIdx.x] = vecLengths[row2];
			//	}
			//}
			//__syncthreads();			
			//int maxRow = shMaxRows[idxR];
			
			int maxRow = vecLengths[row];
			//int maxRow = (int)ceil(vecLengths[row]/(float)(ThreadPerRow*PREFETCH_SIZE) );

			float val[PREFETCH_SIZE];
			int col[PREFETCH_SIZE];
			float dot[PREFETCH_SIZE]={0};

			unsigned int j=0;
			unsigned int arIdx=0;
			for(int i=0; i < maxRow; i++){
				
				#pragma unroll
				for( j=0; j<PREFETCH_SIZE;j++)	{
					//arIdx = (i*PREFETCH_SIZE+j )*align+sliceStart[blockIdx.x]+threadIdx.x;
					arIdx = (i*PREFETCH_SIZE+j )*align+shSliceStart+threadIdx.x;
					col[j] = vecCols[arIdx];
					val[j] = vecVals[arIdx];
				}

				#pragma unroll
				for( j=0; j<PREFETCH_SIZE;j++){
					dot[j] +=val[j]*tex1Dfetch(mainVecTexRef,col[j]); // val[j]* tex1Dfetch(mainVecTexRef,col[j]);
				}
			}

			#pragma unroll
			for( j=1; j<PREFETCH_SIZE;j++){
				dot[0]+=dot[j];	
			}



		shDot[idxT*ROWS_B+idxR]=dot[0];
		__syncthreads();		

		volatile float *shDotv = shDot;
		//reduction to some level
		for( j=blockDim.x/2; j>=ROWS_B; j>>=1) //s/=2
		{
			if(threadIdx.x<j){
				shDotv[threadIdx.x]+=shDotv[threadIdx.x+j];
			}
			__syncthreads();
		}
			
		if(threadIdx.x<ROWS_B){
			//results[row2]=row2;			
			unsigned int row2=blockIdx.x* ROWS_B+threadIdx.x;
			if(row2<nrRows){
			//result[row2]= shDotv[threadIdx.x];
				result[row2]=vecLabels[row2]*shLabel*expf(-shGamma*(selfDot[row2]+shMainSelfDot-2*shDot[threadIdx.x]));
			}
		}

	}//if row<nrRows 
}//end func








/****** nChi2 kernels *******************/
//Use sliced Ellpack format for computing normalized Chi2 kernel, vectors should be histograms
// and normalized according to l1 norm
// K(x,y)= Sum( (xi*yi)/(xi+yi))
//
//vecVals - vectors values in Sliced Ellpack,
//vecCols - array containning column indexes for non zero elements
//vecLengths  - number of non zero elements in row
//sliceStart   - determine where particular slice starts and ends
//result  - for final result
//mainVecIdx - index of main vector
//nrows   - number of rows
//ali	  - align
extern "C" __global__ void nChi2SlEllKernel(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float sh_cache[ThreadPerRow*SliceSize];

		__shared__ int shMainVecIdx;
		__shared__ float shLabel;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shLabel = vecLabels[shMainVecIdx];
		}

		int tx = threadIdx.x;
		int txm = tx % 4; //tx% ThreadPerRow
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);

		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> 2; // 

		if (row < nrRows){
			float sub = 0.0;
			int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
			int col=-1;
			float val1 =0;
			float val2 =0;
			int ind=0;

			for(int i=0; i < maxRow; i++){
				ind = i*align+sliceStart[blockIdx.x]+tx;

				col     = vecCols[ind];
				val1 = vecVals[ind];
				val2 = tex1Dfetch(mainVecTexRef, col);
				sub += (val1*val2)/(val1+val2+FLT_MIN);
			}

			sh_cache[tx] = sub;
			__syncthreads();

			volatile float *shMem = sh_cache;
			//for 4 thread per row

			if(txm < 2){
				shMem[tx]+=shMem[tx+2];
				shMem[tx] += shMem[tx+1];

				if(txm == 0 ){
					result[row]=vecLabels[row]*shLabel*sh_cache[tx];
				}
			}
		}//if row<nrRows  
}//end func



/************* ExpChi2 kernels *******************/

//Use sliced Ellpack format for computing ExpChi2 kernel matrix kolumn
// K(x,y)=exp( -gamma* Sum( (xi-yi)^2/(xi+yi)) =exp(-gamma (sum xi +sum yi -4*sum( (xi*yi)/(xi+yi)) ) )
//vecVals - vectors values in Sliced Ellpack,
//vecCols - array containning column indexes for non zero elements
//vecLengths  - number of non zero elements in row
//sliceStart   - determine where particular slice starts and ends
//selfSum    - precomputed sum of each row
//result  - for final result
//mainVecIdx - index of main vector
//nrows   - number of rows
//ali	  - align
extern "C" __global__ void expChi2SlEllKernel(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfSum,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		__shared__  float sh_cache[ThreadPerRow*SliceSize];

		__shared__ int shMainVecIdx;
		__shared__ float shMainSelfSum;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfSum = selfSum[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
		}

		int tx = threadIdx.x;
		int txm = tx % 4; //tx% ThreadPerRow
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);

		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> 2; // 

		if (row < nrRows){
			float sub = 0.0;
			int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
			int col=-1;
			float val1 =0;
			float val2 =0;
			int ind=0;

			for(int i=0; i < maxRow; i++){
				ind = i*align+sliceStart[blockIdx.x]+tx;

				col     = vecCols[ind];
				val1 = vecVals[ind];
				val2 = tex1Dfetch(mainVecTexRef, col);
				sub += (val1*val2)/(val1+val2+FLT_MIN);
			}

			sh_cache[tx] = sub;
			__syncthreads();

			volatile float *shMem = sh_cache;
			//for 4 thread per row

			if(txm < 2){
				shMem[tx]+=shMem[tx+2];
				shMem[tx] += shMem[tx+1];

				if(txm == 0 ){
					result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfSum[row]+shMainSelfSum-4*sh_cache[tx]));
				}
			}


		}//if row<nrRows 


}//end func


/************************************************************************/
/* 
	Evaluators
*/
/************************************************************************/

extern "C" __global__ void rbfSliceEllpackEvaluator(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* svSelfDot,
	const float* svAlpha,
	const float* svY,
	float * results,
	const int nrRows,
	const int align,
	const float vecSelfDot,
	const float gamma,
	const int texSel)
{

	__shared__ float shGamma;
	__shared__ float shVecSelfDot;
	__shared__ int shRows;
	__shared__  float shDot[ThreadPerRow*SliceSize];
	shDot[threadIdx.x]=0.0;	

	if(threadIdx.x==0)
	{
		shGamma = gamma;
		shVecSelfDot = vecSelfDot,
		shRows= nrRows;
	}
	__syncthreads();

	const int row   = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
	int txm = threadIdx.x %  ThreadPerRow;
	if(row<shRows)
	{
		//hack for choosing different texture reference when launch in different streams
		
		if (texSel==1)
		{
			SpMV_SliceEllpack<1>(vecVals,vecCols,vecLengths,sliceStart,align,row,nrRows,shDot);
		}else{
			SpMV_SliceEllpack<2>(vecVals,vecCols,vecLengths,sliceStart,align,row,nrRows,shDot);
		}

		if(txm == 0 ){
			results[row]=svY[row]*svAlpha[row]*expf(-shGamma*(svSelfDot[row]+shVecSelfDot-2*shDot[threadIdx.x]));
		}

	}	

}



/************************************************************************/
/* 
	Sliced ellpack 
*/
/************************************************************************/

template<int TexSel> __device__ void SpMV_SliceEllpack(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const int align,
	const int row,
	const int nrRows,
	volatile float* shDot)
{
	
	int txm = threadIdx.x %  ThreadPerRow;

	float sub = 0.0;
	int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
	int col=-1;
	float value =0;
	int ind=0;

	for(int i=0; i < maxRow; i++){
		ind = i*align+sliceStart[blockIdx.x]+threadIdx.x;
		col     = vecCols[ind];
		value = vecVals[ind];
		sub += value * fetchTex<TexSel>(col);// tex1Dfetch(mainVecTexRef, col);
	}

	shDot[threadIdx.x] = sub;
	__syncthreads();


	for(int s=ThreadPerRow/2; s>0; s>>=1) //s/=2
	{
		if(txm < s){
			shDot[threadIdx.x] += shDot[threadIdx.x+s];
		}
	}

}


/************************* HELPER FUNCTIONS ****************************/

extern "C" __global__ void makeDenseVectorSlicedEllRT(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	float *mainVector,
	const int mainVecIdx,
	const int nrRows,
	const int vecDim,
	const int align){


		__shared__ int shMaxNNZ;
		__shared__ int shSliceNr;
		__shared__ int shRowInSlice;

		if(threadIdx.x==0)
		{
			shMaxNNZ =	vecLengths[mainVecIdx];
			//in which slice main vector is?
			shSliceNr = mainVecIdx/SliceSize;
			shRowInSlice = mainVecIdx% SliceSize;
		}

		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);
		//int tmx = threadIdx.x % ThreadPerRow;	

		if(thIdx < vecDim)
		{
			//set all vector values to zero
			mainVector[thIdx]=0.0;

			if(thIdx <shMaxNNZ){
				int threadNr = thIdx%ThreadPerRow;
				int rowSlice= thIdx/ThreadPerRow;

				//int	ind = sliceStart[shSliceNr]+shStartRow+tmx;

				int idx = sliceStart[shSliceNr] + align * rowSlice + shRowInSlice * ThreadPerRow + threadNr;

				int col     = vecCols[idx];
				float value = vecVals[idx];
				mainVector[col]=value;
			}


		}//end if

}//end func
