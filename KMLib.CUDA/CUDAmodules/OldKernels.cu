#include "hip/hip_runtime.h"
﻿/*
author: Krzysztof Sopyla
mail: krzysztofsopyla@gmail.com
License: MIT
web page: http://wmii.uwm.edu.pl/~ksopyla/projects/svm-net-with-cuda-kmlib/
*/


//cuda kernel funtion for computing SVM RBF kernel, uses 
// Ellpack-R fromat for storing sparse matrix, labels are in texture cache,  uses ILP - prefetch vector elements in registers
// arrays vals and colIdx should be aligned to PREFETCH_SIZE
//Params:
//vals - array of vectors values
//colIdx  - array of column indexes in ellpack-r fromat
//rowLength -array, contains number of nonzero elements in each row
//selfDot - array of precomputed self linear product 
//results - array of results Linear Kernel
//num_rows -number of vectors
//mainVecIndex - main vector index, needed for retriving its label
//gamma - gamma parameter for RBF 
extern "C" __global__ void rbfEllpackFormatKernel_ILP_old(const float * vals,
									   const int * colIdx, 
									   const int * rowLength, 
									   const float* selfDot,
									   float * results,
									   const int num_rows,
									   const int mainVecIndex,
									   const float gamma)
{
	

	__shared__ float shGamma;
	__shared__ int shMainVecIdx;
	__shared__ float shMainSelfDot;
	__shared__ float shLabel;
	__shared__ int shRows;

	//__shared__ float shDot[PREFETCH_SIZE*BLOCK_SIZE];
	//for(int j=0; j<PREFETCH_SIZE;j++){
	//	shDot[threadIdx.x*PREFETCH_SIZE+j]=0.0;
	//}
			
	//myTex1Dfetch<1>(5);
	
	if(threadIdx.x==0)
	{
		shRows = num_rows;
		shMainVecIdx=mainVecIndex;
		shGamma = gamma;
		shMainSelfDot = selfDot[shMainVecIdx];
		shLabel = tex1Dfetch(labelsTexRef,shMainVecIdx);
	}
	__syncthreads();
		
	const int row   = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index

	if(row<shRows)
	{
		float preVals[PREFETCH_SIZE];
		int preColls[PREFETCH_SIZE];
		//float preVecVals[PREFETCH_SIZE];
		//float dot=0;
		float dot[PREFETCH_SIZE]={0};

		int maxEl = rowLength[row];
	

		for(int i=0; i<maxEl;i++)
		{
			#pragma unroll
			for(int j=0; j<PREFETCH_SIZE;j++)			
			{
				preColls[j]=colIdx[ (i*PREFETCH_SIZE+j)*shRows+row];
				preVals[j]=vals[ (i*PREFETCH_SIZE+j)*shRows+row];
			}
			
			#pragma unroll
			for(int j=0; j<PREFETCH_SIZE;j++){
				//dot+=preVals[j]*tex1Dfetch(mainVecTexRef,preColls[j]);
				dot[j]+=preVals[j]*tex1Dfetch(mainVecTexRef,preColls[j]);
				//shDot[threadIdx.x*PREFETCH_SIZE+j]+=preVals[j]*tex1Dfetch(mainVecTexRef,preColls[j]);
			}
			
		}
		
		
		//volatile float *shMem = shDot;
		//float dot = 0;
		//#pragma unroll
		//for(int j=1; j<PREFETCH_SIZE;j++){
		//		//dot+=shDot[threadIdx.x*PREFETCH_SIZE+j];
		//		shDot[threadIdx.x*PREFETCH_SIZE+0]+=shDot[threadIdx.x*PREFETCH_SIZE+j];
		//}

		//__syncthreads();
		//results[row]=tex1Dfetch(labelsTexRef,row)*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*dot));
		//results[row]=tex1Dfetch(labelsTexRef,row)*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*shDot[threadIdx.x*PREFETCH_SIZE+0]));

		#pragma unroll
		for(int j=1; j<PREFETCH_SIZE;j++){
				dot[0]+=dot[j];
		}
		results[row]=tex1Dfetch(labelsTexRef,row)*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*dot[0]));
		
	}	

}

extern "C" __global__ void rbfEllRTILP_old(const float * vals,
									   const int * colIdx, 
									   const int * rowLength, 
									   const float* selfDot,
									   float * results,
									   const int num_rows,
									   const int mainVecIndex,
									   const float gamma)
{
	

	__shared__ float shGamma;
	__shared__ int shMainVecIdx;
	__shared__ float shMainSelfDot;
	__shared__ float shLabel;
	__shared__ int shRows;

	__shared__ float shDot[BLOCK_SIZE];
	shDot[threadIdx.x]=0.0;	
		
	if(threadIdx.x==0)
	{
		shRows = num_rows;
		shMainVecIdx=mainVecIndex;
		shGamma = gamma;
		shMainSelfDot = selfDot[shMainVecIdx];
		shLabel = tex1Dfetch(labelsTexRef,shMainVecIdx);
	}
	__syncthreads();
		

	//const int idx  = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
	int row  = (blockDim.x * blockIdx.x + threadIdx.x)/THREADS_ROW;

	//const int rowsB= blockDim.x/THREADS_ROW ;//BLOCK_SIZE/THREADS_ROW;  //rows in block
	#define rowsB BLOCK_SIZE/THREADS_ROW

	const int tid = threadIdx.x; // index in block
	const int idxR = tid/THREADS_ROW; //row index mapped into block region
	const int idxT = tid%THREADS_ROW; // thread number in Thread Group

	if(row<shRows)
	{
		float preVals[PREFETCH_SIZE];
		int preColls[PREFETCH_SIZE];
		
		float dot[PREFETCH_SIZE]={0};

		//todo: move to shared mem
		int maxEl = rowLength[row]; //original row length divided by T*PREFETCH

		unsigned int j=0;
		unsigned int arIdx=0;
		
		for(int i=0; i<maxEl;i++)
		{
			
			#pragma unroll
			for( j=0; j<PREFETCH_SIZE;j++)			
			{
				arIdx = (i*PREFETCH_SIZE+j)*shRows*THREADS_ROW+row*THREADS_ROW+idxT;
				preColls[j]=colIdx[arIdx];
				preVals[j]=vals[arIdx];
			}
			
			#pragma unroll
			for( j=0; j<PREFETCH_SIZE;j++){
				dot[j]+=preVals[j]*tex1Dfetch(mainVecTexRef,preColls[j]);
			}
		}

		#pragma unroll
		for( j=1; j<PREFETCH_SIZE;j++){
				dot[0]+=dot[j];
				
		}

		//__syncthreads();	

		// special indexing, values for example for T=4 BlockSize=256
		//for row=0 values are stored on position 0,64,128,192 
		//for row=1 values are stored on position 1,65,129,193 ...
		shDot[idxT*rowsB+idxR]=dot[0];
		
		__syncthreads();		

	
		//reduction to some level
		for( j=blockDim.x/2; j>=rowsB; j>>=1) //s/=2
		{
			if(tid<j){
				shDot[tid]+=shDot[tid+j];
			}
			__syncthreads();
		}			
			
		//if(row2<shRows){
		if(tid<rowsB){
			//results[row2]=row2;			
			unsigned int row2=blockIdx.x* rowsB+tid;
			//results[row2]=shDot[tid];
		    results[row2]=tex1Dfetch(labelsTexRef,row2)*shLabel*expf(-shGamma*(selfDot[row2]+shMainSelfDot-2*shDot[tid]));
		}
	}//if row<nrRows	

}



//cuda kernel funtion for computing SVM RBF kernel, uses 
// Ellpack-R fromat for storing sparse matrix, labels are in texture cache,  uses ILP - prefetch vector elements in registers
//Params:
//vals - array of vectors values
//colIdx  - array of column indexes in ellpack-r fromat
//rowLength -array, contains number of nonzero elements in each row
//selfDot - array of precomputed self linear product 
//results - array of results Linear Kernel
//num_rows -number of vectors
//mainVecIndex - main vector index, needed for retriving its label
//gamma - gamma parameter for RBF 
extern "C" __global__ void rbfEllpackFormatKernel_ILP_shared(const float * vals,
									   const int * colIdx, 
									   const int * rowLength, 
									   const float* selfDot,
									   float * results,
									   const int num_rows,
									   const int mainVecIndex,
									   const float gamma)
{
	

	__shared__ float shGamma;
	__shared__ int shMainVecIdx;
	__shared__ float shMainSelfDot;
	__shared__ float shLabel;

	
	__shared__ float shMainVec[VECDIM];
	
	if(threadIdx.x==0)
	{
		shMainVecIdx=mainVecIndex;
		shGamma = gamma;
		shMainSelfDot = selfDot[shMainVecIdx];
		shLabel = tex1Dfetch(labelsTexRef,shMainVecIdx);
	}

	for(int k=threadIdx.x;k<VECDIM;k+=blockDim.x)
		shMainVec[k]=tex1Dfetch(mainVecTexRef,k);
	
	__syncthreads();
	const int row   = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index

	if(row<num_rows)
	{
		float dot=0;
		int maxEl = rowLength[row];
		
		int i=0;
		
		float preVals[PREFETCH_SIZE];
		int preColls[PREFETCH_SIZE];
		//float preVecVals[PREFETCH_SIZE];
		
		//how many elements are the rest after division
		
		int mainIter = ceilf( (maxEl+0.0)/PREFETCH_SIZE);
		for(i=0; i<mainIter;i++)
		{
			int subIter= min(maxEl-i*PREFETCH_SIZE,PREFETCH_SIZE);
			
			for(int j=0; j<subIter;j++)			
			{
				preColls[j]=colIdx[ (i*PREFETCH_SIZE+j)*num_rows+row];
				preVals[j]=vals[ (i*PREFETCH_SIZE+j)*num_rows+row];
			}			

			for(int j=0; j<subIter;j++){
				dot+=preVals[j]*shMainVec[preColls[j]];
				//dot+=preVals[j]*tex1Dfetch(mainVecTexRef,preColls[j]);
			}
		}
		results[row]=tex1Dfetch(labelsTexRef,row)*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*dot));
		
	}	

}


extern "C" __global__ void rbfEllpackFormatKernel_ILP_sum(const float * vals,
									   const int * colIdx, 
									   const int * rowLength, 
									   const float* selfDot,
									   float * results,
									   const int numRows,
									   const int mainVecIndex,
									   const float gamma)
{
	

	__shared__ float shGamma;
	__shared__ int shMainVecIdx;
	__shared__ float shMainSelfDot;
	__shared__ float shLabel;
	
	if(threadIdx.x==0)
	{
		shMainVecIdx=mainVecIndex;
		shGamma = gamma;
		shMainSelfDot = selfDot[shMainVecIdx];
		shLabel = tex1Dfetch(labelsTexRef,shMainVecIdx);
	}
	__syncthreads();
	
	const int row   = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
	const int num_rows =numRows;
	if(row<num_rows)
	{
		int maxEl = rowLength[row];
		int labelProd = tex1Dfetch(labelsTexRef,row)*shLabel;
		float dot=0;
		
		int col[2]={-1,-1};
		float val[2]={0,0};
		float sum[2] = {0, 0};
		int i=0;
		int sw=0;
		for(i=0; i<maxEl;i++)
		{
			sw = i%2;
			//sw = i&1; //equals i%2
			col[sw]=colIdx[num_rows*i+row];
			val[sw]= vals[num_rows*i+row];
			sum[sw]+=val[sw]*tex1Dfetch(mainVecTexRef,col[sw]);
		}
		dot=sum[0]+sum[1];


		/*int col[PREFETCH_SIZE];
		float val[PREFETCH_SIZE];
		float sum[PREFETCH_SIZE];
		int i=0;
		int sw=0;
		for(i=0; i<maxEl;i++)
		{
			sw = i%PREFETCH_SIZE;
			col[sw]=colIdx[num_rows*i+row];
			val[sw]= vals[num_rows*i+row];
			sum[sw]+=val[sw]*tex1Dfetch(mainVecTexRef,col[sw]);
		}
		for(int k=0; k<PREFETCH_SIZE;k++)
			dot+=sum[k];*/

		results[row]=labelProd*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*dot));
		
	}	

}


extern "C" __global__ void rbfEllpackFormatKernel_old(const float * vals,
									   const int * colIdx, 
									   const int * rowLength, 
									   const float* selfDot,
									   float * results,
									   const int numRows,
									   const int mainVecIndex,
									   const float gamma)
{
	

	__shared__ float shGamma;
	__shared__ int shMainVecIdx;
	__shared__ float shMainSelfDot;
	__shared__ float shLabel;
	__shared__ int shRows;
	
	if(threadIdx.x==0)
	{
		shMainVecIdx=mainVecIndex;
		shGamma = gamma;
		shMainSelfDot = selfDot[shMainVecIdx];
		shLabel = tex1Dfetch(labelsTexRef,shMainVecIdx);
		shRows=numRows;
	}
	__syncthreads();
	
	const int row   = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
	const int num_rows =numRows;
	if(row<shRows)
	{
		
		int maxEl = rowLength[row];
		int labelProd = tex1Dfetch(labelsTexRef,row)*shLabel;
		float dot=0;
		
		int col=-1;
		float val=0;
		int i=0;
		for(i=0; i<maxEl;i++)
		{
			col=colIdx[num_rows*i+row];
			val= vals[num_rows*i+row];
			dot+=val*tex1Dfetch(mainVecTexRef,col);
		}

		
		//float dot = SpMV_Ellpack<1>(vals,colIdx,rowLength,row,numRows);
		results[row]=labelProd*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*dot));;
	}	

}


/************ Sliced Ellpack *********************/
//Use sliced Ellpack format for computing rbf kernel
//vecVals - vectors values in Sliced Ellpack,
//vecCols - array containning column indexes for non zero elements
//vecLengths  - number of non zero elements in row
//sliceStart   - determine where particular slice starts and ends
//selfDot    - precomputed self dot product
//result  - for final result
//mainVecIdx - index of main vector
//nrows   - number of rows
//ali	  - align
extern "C" __global__ void rbfSlicedEllpackKernel_old(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfDot,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float sh_cache[ThreadPerRow*SliceSize];

		__shared__ int shMainVecIdx;
		__shared__ float shMainSelfDot;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfDot = selfDot[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
		}

		int tx = threadIdx.x;
		int txm = tx %  ThreadPerRow;
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);

		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> LOG_THREADS; // 

		if (row < nrRows){
			float sub = 0.0;
			int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
			int col=-1;
			float value =0;
			int ind=0;

			for(int i=0; i < maxRow; i++){
				ind = i*align+sliceStart[blockIdx.x]+tx;
				col     = vecCols[ind];
				value = vecVals[ind];
				sub += value * tex1Dfetch(mainVecTexRef, col);
			}

			sh_cache[tx] = sub;
			__syncthreads();

			volatile float *shMem = sh_cache;


			for(int s=ThreadPerRow/2; s>0; s>>=1) //s/=2
			{
				if(txm < s){
					shMem[tx] += shMem[tx+s];
				}
			}

			if(txm == 0 ){
				result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
			}


			//for 4 thread per row
			//if(txm < 2){
			//	shMem[tx]+=shMem[tx+2];
			//	shMem[tx] += shMem[tx+1];
			//if(txm < 1){
			//	shMem[tx] += shMem[tx+1];
			//	if(txm == 0 ){
			//		result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
			//	}
			//}
		}//if row<nrRows 
}//end func



extern "C" __global__ void rbfSlicedEllpackKernel_shared(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfDot,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float sh_cache[ThreadPerRow*SliceSize];

		__shared__ int shMainVecIdx;
		__shared__ float shMainSelfDot;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfDot = selfDot[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
		}

		__shared__ float shMainVecAR[VECDIM];
		volatile float *shMainVec =shMainVecAR;

		for(int k=threadIdx.x;k<VECDIM;k+=blockDim.x)
			shMainVec[k]=tex1Dfetch(mainVecTexRef,k);

		__syncthreads();

		int tx = threadIdx.x;
		int txm = tx % 4; //tx% ThreadPerRow
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);

		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> 2; // 

		if (row < nrRows){
			float sub = 0.0;
			int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
			int labelProd = vecLabels[row]*shLabel;
			int ind = -1;
			int col =-1;
			float value=0;

			for(int i=0; i < maxRow; i++){
				ind = i*align+sliceStart[blockIdx.x]+tx;

				col     = vecCols[ind];
				value = vecVals[ind];

				sub += value * shMainVec[col];
			}

			sh_cache[tx] = sub;
			__syncthreads();

			volatile float *shMem = sh_cache;
			//for 4 thread per row

			if(txm < 2){
				shMem[tx]+=shMem[tx+2];
				shMem[tx] += shMem[tx+1];

				if(txm == 0 ){
					//result[row]=vecLabels[row]*shLabel*expf(-0.5*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
					value = labelProd*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
					//value = floorf(value*1000+0.5)/1000;
					result[row]=value;
					//result[row]=vecLabels[row]*shLabel*expf(-GammaDev*(selfDot[row]+shMainSelfDot-2*sh_cache[tx]));
				}
			}


		}//if row<nrRows 



}//end func



extern "C" __global__ void rbfSERTILP_old(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfDot,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float shDot[ThreadPerRow*SliceSize];

		//define ROWS_B BLOCK_SIZE/THREADS_ROW
#define ROWS_B SliceSize
		//__shared__ int shMaxRows[ROWS_B];

		__shared__ int shMainVecIdx;
		__shared__ int shSliceStart;
		__shared__ float shMainSelfDot;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfDot = selfDot[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
			shSliceStart=sliceStart[blockIdx.x];
		}
		__syncthreads();

		//int tid = threadIdx.x;

		int idxT = threadIdx.x % ThreadPerRow; //thread number in Thread group
		int idxR = threadIdx.x/ThreadPerRow; //row index mapped into block region

		//int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);
		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  (blockIdx.x*blockDim.x+threadIdx.x)>> LOG_THREADS; 

		if (row < nrRows){


			//if(threadIdx.x < ROWS_B){
			//	unsigned int row2=blockIdx.x* ROWS_B+threadIdx.x;
			//	if(row2<nrRows){
			//		shMaxRows[threadIdx.x] = vecLengths[row2];
			//	}
			//}
			//__syncthreads();			
			//int maxRow = shMaxRows[idxR];

			int maxRow = vecLengths[row];
			//int maxRow = (int)ceil(vecLengths[row]/(float)(ThreadPerRow*PREFETCH_SIZE) );

			float val[PREFETCH_SIZE];
			int col[PREFETCH_SIZE];
			float dot[PREFETCH_SIZE]={0};

			unsigned int j=0;
			unsigned int arIdx=0;
			for(int i=0; i < maxRow; i++){

#pragma unroll
				for( j=0; j<PREFETCH_SIZE;j++)	{
					//arIdx = (i*PREFETCH_SIZE+j )*align+sliceStart[blockIdx.x]+threadIdx.x;
					arIdx = (i*PREFETCH_SIZE+j )*align+shSliceStart+threadIdx.x;
					col[j] = vecCols[arIdx];
					val[j] = vecVals[arIdx];
				}

#pragma unroll
				for( j=0; j<PREFETCH_SIZE;j++){
					dot[j] +=val[j]*tex1Dfetch(mainVecTexRef,col[j]); 
				}
			}

#pragma unroll
			for( j=1; j<PREFETCH_SIZE;j++){
				dot[0]+=dot[j];	
			}



			shDot[idxT*ROWS_B+idxR]=dot[0];
			__syncthreads();		

			volatile float *shDotv = shDot;
			//reduction to some level
			for( j=blockDim.x/2; j>=ROWS_B; j>>=1) //s/=2
			{
				if(threadIdx.x<j){
					shDotv[threadIdx.x]+=shDotv[threadIdx.x+j];
				}
				__syncthreads();
			}

			if(threadIdx.x<ROWS_B){
				//results[row2]=row2;			
				unsigned int row2=blockIdx.x* ROWS_B+threadIdx.x;
				if(row2<nrRows){
					//result[row2]= shDotv[threadIdx.x];
					result[row2]=vecLabels[row2]*shLabel*expf(-shGamma*(selfDot[row2]+shMainSelfDot-2*shDot[threadIdx.x]));
				}
			}

		}//if row<nrRows 
}//end func





extern "C" __global__ void nChi2EllpackKernel(const float * vals,
	const int * colIdx, 
	const int * rowLength, 
	float * results,
	const int numRows,
	const int mainVecIndex)
{

	__shared__ float shLabel;
	__shared__ int shRows;

	if(threadIdx.x==0)
	{
		shLabel = tex1Dfetch(labelsTexRef,mainVecIndex);	
		shRows=numRows;
	}

	__syncthreads();

	const int row   = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
	const int num_rows =numRows;
	if(row<shRows)
	{
		int maxEl = rowLength[row];
		float labelProd = tex1Dfetch(labelsTexRef,row)*shLabel;
		float chi=0;

		int col1=-1;
		float val1=0;
		float val2=0;
		int i=0;

		for(i=0; i<maxEl;i++)
		{
			col1=colIdx[num_rows*i+row];
			val1= vals[num_rows*i+row];
			val2 = tex1Dfetch(mainVecTexRef,col1);

			chi+= (val1*val2)/(val1+val2+FLT_MIN);

		}
		results[row]=labelProd*chi;
	}	

}

extern "C" __global__ void nChi2EllRTILP(const float * vals,
	const int * colIdx, 
	const int * rowLength, 
	float * results,
	const int num_rows,
	const int mainVecIndex)
{



	__shared__ int shMainVecIdx;
	__shared__ float shLabel;
	__shared__ int shRows;

	__shared__ float shChi2[BLOCK_SIZE];
	shChi2[threadIdx.x]=0.0;	

	if(threadIdx.x==0)
	{
		shRows = num_rows;
		shMainVecIdx=mainVecIndex;
		shLabel = tex1Dfetch(labelsTexRef,shMainVecIdx);
	}
	__syncthreads();


	//const int idx  = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
	int row  = (blockDim.x * blockIdx.x + threadIdx.x)/THREADS_ROW;

	const int rowsB= blockDim.x/THREADS_ROW ;//BLOCK_SIZE/THREADS_ROW;  //rows in block
	//#define rowsB (BLOCK_SIZE/THREADS_ROW)

	const int tid = threadIdx.x; // index in block
	const int idxR = tid/THREADS_ROW; //row index mapped into block region
	const int idxT = tid%THREADS_ROW; // thread number in Thread Group

	if(row<shRows)
	{
		float vals[PREFETCH_SIZE];
		float val2=0;
		int cols[PREFETCH_SIZE];

		float dot[PREFETCH_SIZE]={0};

		int maxEl = rowLength[row]; //original row length divided by T*PREFETCH

		unsigned int j=0;
		unsigned int arIdx=0;

		for(int i=0; i<maxEl;i++)
		{

#pragma unroll
			for( j=0; j<PREFETCH_SIZE;j++)			
			{
				arIdx = (i*PREFETCH_SIZE+j)*shRows*THREADS_ROW+row*THREADS_ROW+idxT;
				cols[j]=colIdx[arIdx];
				vals[j]=vals[arIdx];
			}

#pragma unroll
			for( j=0; j<PREFETCH_SIZE;j++){
				val2=tex1Dfetch(mainVecTexRef,cols[j]);
				dot[j]+=(vals[j]*val2)/(vals[j]+val2+FLT_MIN);
			}

		}


#pragma unroll
		for( j=1; j<PREFETCH_SIZE;j++){
			dot[0]+=dot[j];

		}

		//__syncthreads();	

		// special indexing, values for example for T=4 BlockSize=256
		//for row=0 values are stored on position 0,64,128,192 
		//for row=1 values are stored on position 1,65,129,193 ...
		shChi2[idxT*rowsB+idxR]=dot[0];

		__syncthreads();		


		//reduction to some level
		for( j=blockDim.x/2; j>=rowsB; j>>=1) //s/=2
		{
			if(tid<j){
				shChi2[tid]+=shChi2[tid+j];
			}
			__syncthreads();
		}			

		//if(row2<shRows){
		if(tid<rowsB){
			//results[row2]=row2;			
			unsigned int row2=blockIdx.x* rowsB+tid;
			//results[row2]=shDot[tid];
			results[row2]=shChi2[tid];
		}
	}	

}


	extern "C" __global__ void nChi2EllpackKernel_old(const float * vals,
		const int * colIdx, 
		const int * rowLength, 
		float * results,
		const int numRows,
		const int mainVecIndex)
	{

		__shared__ float shLabel;
		__shared__ int shRows;

		if(threadIdx.x==0)
		{
			shLabel = tex1Dfetch(labelsTexRef,mainVecIndex);	
			shRows=numRows;
		}

		__syncthreads();

		const int row   = blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
		const int num_rows =numRows;
		if(row<shRows)
		{
			int maxEl = rowLength[row];
			float labelProd = tex1Dfetch(labelsTexRef,row)*shLabel;
			float chi=0;

			int col1=-1;
			float val1=0;
			float val2=0;
			int i=0;

			for(i=0; i<maxEl;i++)
			{
				col1=colIdx[num_rows*i+row];
				val1= vals[num_rows*i+row];
				val2 = tex1Dfetch(mainVecTexRef,col1);

				chi+= (val1*val2)/(val1+val2+FLT_MIN);

			}
			results[row]=labelProd*chi;
		}	

	}


	//summary: cuda kernel for evaluation, predicts new unseen elements using linear SVM kernel,
	// first elements matrix is in sparse CSR format, second (support vectors) matrix B is 
	// in column major order (each kolumn is in dense format, in 'svTexRef' texture cache)
	// you have to Launch this kernel as many times as support vectors, each time
	// copy new support vector into texture cache
	//params:
	//AVals - values for first matrix
	//AIdx - indexes for first matrix
	//APtrs - pointers to next vector
	//svLabels - support vector labels
	//svAlphas - support vector alphas coef 
	//result - result matrix
	//ARows - number of rows in first matrix
	//BCols - number of cols in second matrix
	//ColumnIndex - index of support vector in B matrix
	extern "C" __global__ void linearCSREvaluatorDenseVector(const float * AVals,
		const int * AIdx, 
		const int * APtrs, 
		const float * svLabels,
		const float * svAlphas,
		float * result,
		const int ARows,
		const int BCols,
		const int ColumnIndex)
	{
		__shared__ float sdata[BLOCK_SIZE + 16];                          // padded to avoid reduction ifs
		__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];

		const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
		const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
		const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
		const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
		const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

		for(int row = warp_id; row < ARows; row += num_warps){
			// use two threads to fetch Ap[row] and Ap[row+1]
			// this is considerably faster than the straightforward version
			if(thread_lane < 2)
				ptrs[warp_lane][thread_lane] = APtrs[row + thread_lane];
			const int row_start = ptrs[warp_lane][0];                   //same as: row_start = Ap[row];
			const int row_end   = ptrs[warp_lane][1];                   //same as: row_end   = Ap[row+1];

			// compute local sum
			float sum = 0;
			for(int jj = row_start + thread_lane; jj < row_end; jj += WARP_SIZE)
				sum += AVals[jj] * tex1Dfetch(svTexRef,AIdx[jj]);

			// reduce local sums to row sum (ASSUME: warpsize 32)
			sdata[threadIdx.x] = sum;
			sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads(); 
			sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
			sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
			sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
			sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();



			// first thread writes warp result
			if (thread_lane == 0)
			{
				//remember that we use result memory for storing partial result
				//so the size of array is the same as number of elements
				result[row]+=sdata[threadIdx.x]*svLabels[ColumnIndex]*svAlphas[ColumnIndex];
				//row major order
				//result[row*BCols+ColumnIndex]= sdata[threadIdx.x];
				//column major order
				//result[ColumnIndex*ARows+row]= sdata[threadIdx.x];
			}


		}
	}

	//cuda kernel function for computing SVM RBF kernel, uses 
// CSR format for storing sparse matrix, labels are in texture cache, 
//Remarks: based on spmv_csr_vector_kernel from publication above
//Params:
//vals - array of vectors values
//idx  - array of vectors indexes in CSR format
//vecPointers -array of pointers(indexes) to idx and vals array to specific vectors
//selfDot - array of precomputed self linear product 
//results - array of results Linear Kernel
//num_rows -number of vectors, stored in CSR matrix format, each vector is stored in one row of matrix
//mainVecIndex - main vector index, needed for retrieving its label
//gamma - gamma parameter for RBF 
extern "C" __global__ void rbfCsrFormatKernel(const float * vals,
									   const int * idx, 
									   const int * vecPointers, 
									   const float* selfDot,
									   float * results,
									   const int num_rows,
									   const int mainVecIndex,
									   const float gamma)
{
	__shared__ float sdata[BLOCK_SIZE + 16];                    // padded to avoid reduction ifs
	__shared__ int ptrs[BLOCK_SIZE/WARP_SIZE][2];
	__shared__ float shGamma;
	__shared__ int shMainVecIdx;
	__shared__ float shMainSelfDot;
	__shared__ float shLabel;
	
	if(threadIdx.x==0)
	{
		shMainVecIdx=mainVecIndex;
		shGamma = gamma;
		shMainSelfDot = selfDot[shMainVecIdx];
		shLabel = tex1Dfetch(labelsTexRef,shMainVecIdx);
	}	
	const int thread_id   = BLOCK_SIZE * blockIdx.x + threadIdx.x;  // global thread index
	const int thread_lane = threadIdx.x & (WARP_SIZE-1);            // thread index within the warp
	const int warp_id     = thread_id   / WARP_SIZE;                // global warp index
	const int warp_lane   = threadIdx.x / WARP_SIZE;                // warp index within the CTA
	const int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // total number of active warps

	for(int row = warp_id; row < num_rows; row += num_warps){
		// use two threads to fetch vecPointers[row] and vecPointers[row+1]
		// this is considerably faster than the straightforward version
		if(thread_lane < 2)
			ptrs[warp_lane][thread_lane] = vecPointers[row + thread_lane];
		const int row_start = ptrs[warp_lane][0];            //same as: row_start = vecPointers[row];
		const int row_end   = ptrs[warp_lane][1];            //same as: row_end   = vecPointers[row+1];

		// compute local sum
		float sum = 0;
		for(int jj = row_start + thread_lane; jj < row_end; jj += WARP_SIZE)
		{
			sum += vals[jj] * tex1Dfetch(mainVecTexRef,idx[jj]);
			//__syncthreads();
		}

		// reduce local sums to row sum (ASSUME: warpsize 32)
/*		 old code not working on fermi card
sdata[threadIdx.x] = sum;
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x + 16]; __syncthreads(); 
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  8]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  4]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  2]; __syncthreads();
		sdata[threadIdx.x] = sum = sum + sdata[threadIdx.x +  1]; __syncthreads();
*/
		volatile float* smem = sdata;
		smem[threadIdx.x] = sum; __syncthreads(); 
		smem[threadIdx.x] = sum = sum + smem[threadIdx.x + 16]; //__syncthreads(); 
		smem[threadIdx.x] = sum = sum + smem[threadIdx.x +  8]; //__syncthreads();
		smem[threadIdx.x] = sum = sum + smem[threadIdx.x +  4]; //__syncthreads();
		smem[threadIdx.x] = sum = sum + smem[threadIdx.x +  2]; //__syncthreads();
		smem[threadIdx.x] = sum = sum + smem[threadIdx.x +  1]; //__syncthreads();

		// first thread writes warp result
		if (thread_lane == 0){
			//results[row]=tex1Dfetch(labelsTexRef,row)*tex1Dfetch(labelsTexRef,shMainVecIdx)*expf(-shGamma*(selfDot[row]+selfDot[shMainVecIdx]-2*sdata[threadIdx.x]));
			
			results[row]=tex1Dfetch(labelsTexRef,row)*shLabel*expf(-shGamma*(selfDot[row]+shMainSelfDot-2*smem[threadIdx.x]));
		}
	}
}

extern "C" __global__ void nChi2SlEllKernel(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const int align){

		//sh_data size = SliceSize*ThreadsPerRow*sizeof(float)
		//float* sh_cache = (float*)sh_data;
		__shared__  float sh_cache[ThreadPerRow*SliceSize];

		__shared__ int shMainVecIdx;
		__shared__ float shLabel;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shLabel = vecLabels[shMainVecIdx];
		}

		int tx = threadIdx.x;
		int txm = tx % 4; //tx% ThreadPerRow
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);

		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> 2; // 

		if (row < nrRows){
			float sub = 0.0;
			int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
			int col=-1;
			float val1 =0;
			float val2 =0;
			int ind=0;

			for(int i=0; i < maxRow; i++){
				ind = i*align+sliceStart[blockIdx.x]+tx;

				col     = vecCols[ind];
				val1 = vecVals[ind];
				val2 = tex1Dfetch(mainVecTexRef, col);
				sub += (val1*val2)/(val1+val2+FLT_MIN);
			}

			sh_cache[tx] = sub;
			__syncthreads();

			volatile float *shMem = sh_cache;
			//for 4 thread per row

			if(txm < 2){
				shMem[tx]+=shMem[tx+2];
				shMem[tx] += shMem[tx+1];

				if(txm == 0 ){
					result[row]=vecLabels[row]*shLabel*sh_cache[tx];
				}
			}
		}//if row<nrRows  
}//end func

extern "C" __global__ void expChi2SlEllKernel(const float *vecVals,
	const int *vecCols,
	const int *vecLengths, 
	const int * sliceStart, 
	const float* selfSum,
	const float* vecLabels,
	float *result,
	const int mainVecIdx,
	const int nrRows,
	const float gamma, 
	const int align){

		__shared__  float sh_cache[ThreadPerRow*SliceSize];

		__shared__ int shMainVecIdx;
		__shared__ float shMainSelfSum;
		__shared__ float shLabel;
		__shared__ float shGamma;

		if(threadIdx.x==0)
		{
			shMainVecIdx=mainVecIdx;
			shMainSelfSum = selfSum[shMainVecIdx];
			shLabel = vecLabels[shMainVecIdx];
			shGamma=gamma;
		}

		int tx = threadIdx.x;
		int txm = tx % 4; //tx% ThreadPerRow
		int thIdx = (blockIdx.x*blockDim.x+threadIdx.x);

		//map group of thread to row, in this case 4 threads are mapped to one row
		int row =  thIdx>> 2; // 

		if (row < nrRows){
			float sub = 0.0;
			int maxRow = (int)ceil(vecLengths[row]/(float)ThreadPerRow);
			int col=-1;
			float val1 =0;
			float val2 =0;
			int ind=0;

			for(int i=0; i < maxRow; i++){
				ind = i*align+sliceStart[blockIdx.x]+tx;

				col     = vecCols[ind];
				val1 = vecVals[ind];
				val2 = tex1Dfetch(mainVecTexRef, col);
				sub += (val1*val2)/(val1+val2+FLT_MIN);
			}

			sh_cache[tx] = sub;
			__syncthreads();

			volatile float *shMem = sh_cache;
			//for 4 thread per row

			if(txm < 2){
				shMem[tx]+=shMem[tx+2];
				shMem[tx] += shMem[tx+1];

				if(txm == 0 ){
					result[row]=vecLabels[row]*shLabel*expf(-shGamma*(selfSum[row]+shMainSelfSum-4*sh_cache[tx]));
				}
			}


		}//if row<nrRows 


}//end func